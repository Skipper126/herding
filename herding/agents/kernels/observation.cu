#include "hip/hip_runtime.h"
#define DOG 0
#define SHEEP 1
#define DEG2RAD 0.01745329252

__device__ void clear_observation(Arrays *arrays)
{
    for (int i = 0; i < 2; ++i)
    {
        for (int j = 0; j < 3; ++j)
        {
            arrays->observation[threadIdx.x][threadIdx.y][i][j] = 0.5;
        }
    }
    arrays->rays_lengths[threadIdx.x][threadIdx.y] = 1;
}

__device__ float get_distance(float x1, float y1, float x2, float y2)
{
    float x_diff = x1 - x2;
    float y_diff = y1 - y2;
    return sqrtf((x_diff * x_diff) + (y_diff * y_diff));
}

__global__ void get_observation(Arrays *arrays)
{
    int dog_index = threadIdx.x;
    int ray_index = threadIdx.y;
    clear_observation(arrays);
    float dog_pos_x = arrays->dogs_positions[dog_index][0];
    float dog_pos_y = arrays->dogs_positions[dog_index][1];
    float ray_angle = arrays->dogs_rotations[dog_index] + (((float)threadIdx.y / RAYS_COUNT) * PI);
    float min_distance = RAY_LENGTH;
    if (ray_angle > 2 * PI)
    {
        ray_angle = ray_angle - 2 * PI;
    }
    for (int i = 0; i < SHEEP_COUNT; ++i)
    {
        float agent_pos_x = arrays->sheep_positions[i][0];
        float agent_pos_y = arrays->sheep_positions[i][1];
        float distance = get_distance(dog_pos_x, dog_pos_y, agent_pos_x, agent_pos_y);
        
        if (distance < min_distance)
        {
            float angle = (atan2f(dog_pos_y - agent_pos_y, dog_pos_x - agent_pos_x) + PI);

            if (fabsf(angle - ray_angle) < atanf(AGENT_RADIUS / distance))
            {
                min_distance = distance;
                arrays->observation[dog_index][ray_index][0][0] = SHEEP_COLOR_R;
                arrays->observation[dog_index][ray_index][0][1] = SHEEP_COLOR_G;
                arrays->observation[dog_index][ray_index][0][2] = SHEEP_COLOR_B;
                arrays->rays_lengths[dog_index][ray_index] = distance / RAY_LENGTH;
            }
        }
    }

    for (int i = 0; i < DOGS_COUNT; ++i)
    {
        if (i == dog_index)
            continue;

        float agent_pos_x = arrays->dogs_positions[i][0];
        float agent_pos_y = arrays->dogs_positions[i][1];
        float distance = get_distance(dog_pos_x, dog_pos_y, agent_pos_x, agent_pos_y);

        if (distance < min_distance)
        {
            float angle = (atan2f(dog_pos_y - agent_pos_y, dog_pos_x - agent_pos_x) + PI);

            if (fabsf(angle - ray_angle) < atanf(AGENT_RADIUS / distance))
            {
                min_distance = distance;
                arrays->observation[dog_index][ray_index][0][0] = DOG_COLOR_R;
                arrays->observation[dog_index][ray_index][0][1] = DOG_COLOR_G;
                arrays->observation[dog_index][ray_index][0][2] = DOG_COLOR_B;
                arrays->rays_lengths[dog_index][ray_index] = distance / RAY_LENGTH;
            }
        }
    }

}
